
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <sys/time.h>

#define N 1024	// Only powers of 2 to simplify the code
#define BLOCK_SIZE 32
#define NUM_BLOCKS N
#define NUM_THREADS_PER_BLOCK N
#define NUM_BLOCKS_TILED (N*N)/(BLOCK_SIZE*BLOCK_SIZE)
#define NUM_THREADS_PER_BLOCK_TILED BLOCK_SIZE*BLOCK_SIZE
#define TIME_RESOLUTION 1000000

using namespace std;

long long unsigned initial_time;
struct timeval t;

void printResults (long long unsigned tt) {
	cout << "Execution time: " << tt << " usecs" << endl;
}

void start (void) {
	gettimeofday(&t, NULL);
	initial_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;
}

long long unsigned stop (void) {
	gettimeofday(&t, NULL);
	long long unsigned final_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;

	return final_time - initial_time;
}

__global__ void matrixKernel (float **dev_m1, float **dev_m2, float **dev_result) {

    dev_result[blockIdx.x][threadIdx.x]=0;

	for(unsigned i=0; i < N; i++)
		dev_result[blockIdx.x][threadIdx.x] += dev_m1[blockIdx.x][i]*dev_m2[i][threadIdx.x];
}

void gpuMatrixMult (float **m1, float **m2, float **result) {
	float **dev_m1, **dev_m2, **dev_result;

	hipMalloc((void***) &dev_m1, N * sizeof(float *));
	hipMalloc((void***) &dev_m2, N * sizeof(float *));
	hipMalloc((void***) &dev_result, N * sizeof(float *));

	for(unsigned i=0; i < N; i++){
		hipMalloc((void**) &(dev_m1[i]), N * sizeof(float));
		hipMalloc((void**) &(dev_m2[i]), N * sizeof(float));
		hipMalloc((void**) &(dev_result[i]), N * sizeof(float));
	}
    
	//startTime
	start();

	hipMemcpy(dev_m1, m1, N * sizeof(float *), hipMemcpyHostToDevice);
	hipMemcpy(dev_m2, m2, N * sizeof(float *), hipMemcpyHostToDevice);
	
	for(unsigned i=0; i < N; i++){
		hipMemcpy(dev_m1[i], m1[i], N * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_m2[i], m2[i], N * sizeof(float), hipMemcpyHostToDevice);
	}

	matrixKernel <<< NUM_THREADS_PER_BLOCK, NUM_BLOCKS >>>(dev_m1, dev_m2, dev_result);

	// copy the output to the host
	hipMemcpy(result, dev_result, N * sizeof(float *), hipMemcpyDeviceToHost);
	for(unsigned i=0; i < N; i++)
		hipMemcpy(result[i], dev_result[i], N * sizeof(float), hipMemcpyDeviceToHost);

	//stopTime
    long long unsigned time = stop();
	printResults(time);

	// free the device memory
	hipFree(dev_m1);
	hipFree(dev_m2);
	hipFree(dev_result);
}


__global__ void tiledMatrixKernel (float **dev_m1, float **dev_m2, float **dev_result) {

	__shared__ float temp1 [BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float temp2 [BLOCK_SIZE][BLOCK_SIZE];
	int xIn = threadIdx.x/BLOCK_SIZE;
	int yIn = threadIdx.x%BLOCK_SIZE;
	int xB = (((blockIdx.x*BLOCK_SIZE) / N) * BLOCK_SIZE) + xIn;
	int yB = (((blockIdx.x*BLOCK_SIZE) % N) * BLOCK_SIZE) + yIn;

	temp1[xIn][yIn]=dev_m1[xB][yB];
	temp2[xIn][yIn]=dev_m2[xB][yB];

	__syncthreads();

	for(unsigned i=0; i < BLOCK_SIZE; i++)
		dev_result[xB][yB] += temp1[xIn][i]*temp2[i][yIn];
}

void gpuTiledMatrixMult (float **m1, float **m2, float **result) {
	float **dev_m1, **dev_m2, **dev_result;

	hipMalloc((void***) &dev_m1, N * sizeof(float *));
	hipMalloc((void***) &dev_m2, N * sizeof(float *));
	hipMalloc((void***) &dev_result, N * sizeof(float *));

	for(unsigned i=0; i < N; i++){
		hipMalloc((void**) &(dev_m1[i]), N * sizeof(float));
		hipMalloc((void**) &(dev_m2[i]), N * sizeof(float));
		hipMalloc((void**) &(dev_result[i]), N * sizeof(float));
	}
    
    start();
	//startKernelTime();

	hipMemcpy(dev_m1, m1, N * sizeof(float *), hipMemcpyHostToDevice);
	hipMemcpy(dev_m2, m2, N * sizeof(float *), hipMemcpyHostToDevice);
	hipMemcpy(dev_result, result, N * sizeof(float *), hipMemcpyHostToDevice);
	
	for(unsigned i=0; i < N; i++){
		hipMemcpy(dev_m1[i], m1[i], N * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_m2[i], m2[i], N * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_result[i], result[i], N * sizeof(float), hipMemcpyHostToDevice);
	}

	tiledMatrixKernel <<< NUM_THREADS_PER_BLOCK_TILED, NUM_BLOCKS_TILED >>>(dev_m1, dev_m2, dev_result);

	// copy the output to the host
	hipMemcpy(result, dev_result, N * sizeof(float *), hipMemcpyDeviceToHost);
	for(unsigned i=0; i < N; i++)
		hipMemcpy(result[i], dev_result[i], N * sizeof(float), hipMemcpyDeviceToHost);

	//stopKernelTime();
    long long unsigned time = stop();
	printResults(time);

	// free the device memory
	hipFree(dev_m1);
	hipFree(dev_m2);
	hipFree(dev_result);
}

int main (int argc, char *argv[]) {

    unsigned seed=0;
    float **a = (float**)malloc(sizeof(float*)*N);
    float **b = (float**)malloc(sizeof(float*)*N);
    float **c = (float**)malloc(sizeof(float*)*N);

    srand(seed);

    //build matrix A with random values and C initilized with 0's
    for(int i = 0; i < N; i++){
        c[i] = (float*) malloc(sizeof(float)*N);
        a[i] = (float*) malloc(sizeof(float)*N);
        for(int j = 0; j < N; j++){
            a[i][j] = rand();
            c[i][j] = 0;
        }   
    }

    //build matrix B with all elements equals to 1
    for(int i = 0; i < N; i++){
        b[i] = (float*) malloc(sizeof(float)*N);
        for(int j = 0; j < N; j++)
            b[i][j] = 1;
    }

	gpuMatrixMult(a,b,c);

	for (unsigned i = 0; i < N; i++)
		for (unsigned j = 0; j < N; j++)
			c[i][j] = 0;

	gpuTiledMatrixMult(a,b,c);

	return 0;
}
