
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <sys/time.h>

#define N 32	// Only powers of 2 to simplify the code
#define BLOCK_SIZE 8
#define NUM_BLOCKS N
#define NUM_THREADS_PER_BLOCK N
#define NUM_BLOCKS_TILED (N*N)/(BLOCK_SIZE*BLOCK_SIZE)
#define NUM_THREADS_PER_BLOCK_TILED BLOCK_SIZE*BLOCK_SIZE
#define TIME_RESOLUTION 1000000

using namespace std;

long long unsigned initial_time;
struct timeval t;

void printResults (long long unsigned tt) {
	cout << tt << endl;
}

void start (void) {
	gettimeofday(&t, NULL);
	initial_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;
}

long long unsigned stop (void) {
	gettimeofday(&t, NULL);
	long long unsigned final_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;

	return final_time - initial_time;
}

__global__ void matrixKernel (float *dev_m1, float *dev_m2, float *dev_result) {
    
    *(dev_result+blockIdx.x*N+threadIdx.x)=0;
    
	for(unsigned i=0; i < N; i++)
		*(dev_result+blockIdx.x*N+threadIdx.x) += *(dev_m1+blockIdx.x*N+i) * *(dev_m2+i*N+threadIdx.x);
}

void gpuMatrixMult (float *m1, float *m2, float *result) {
	float *dev_m1, *dev_m2, *dev_result;

	hipMalloc((void**) &dev_m1,N * N * sizeof(float));
	hipMalloc((void**) &dev_m2,N * N * sizeof(float));
	hipMalloc((void**) &dev_result, N * N * sizeof(float));
    
	//startTime
	start();
	
    hipMemcpy(dev_m1, m1, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_m2, m2, N * N * sizeof(float), hipMemcpyHostToDevice);

	matrixKernel <<< NUM_THREADS_PER_BLOCK, NUM_BLOCKS >>>(dev_m1, dev_m2, dev_result);
    
	// copy the output to the host
    hipMemcpy(result, dev_result, N * N * sizeof(float), hipMemcpyDeviceToHost);
    
	//stopTime
	printResults(stop());

	// free the device memory
	hipFree(dev_m1);
	hipFree(dev_m2);
	hipFree(dev_result);
}

/*
__global__ void tiledMatrixKernel (float *dev_m1, float *dev_m2, float *dev_result) {

	__shared__ float temp1 [BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float temp2 [BLOCK_SIZE][BLOCK_SIZE];
	int xIn = threadIdx.x/BLOCK_SIZE;
	int yIn = threadIdx.x%BLOCK_SIZE;
	int xB = (((blockIdx.x*BLOCK_SIZE) / N) * BLOCK_SIZE) + xIn;
	int yB = (((blockIdx.x*BLOCK_SIZE) % N) * BLOCK_SIZE) + yIn;

	temp1[xIn][yIn]=*(dev_m1+xB*N+yB);
	temp2[xIn][yIn]=*(dev_m2+xB*N+yB);

	__syncthreads();

	for(unsigned i=0; i < BLOCK_SIZE; i++)
		*(dev_result+xB*N+yB) += temp1[xIn][i]*temp2[i][yIn];
}

void gpuTiledMatrixMult (float *m1, float *m2, float *result) {
	float *dev_m1, *dev_m2, *dev_result;

	cudaMalloc((void**) &dev_m1,N * N * sizeof(float));
	cudaMalloc((void**) &dev_m2,N * N * sizeof(float));
	cudaMalloc((void**) &dev_result, N * N * sizeof(float));
    
    start();
	//startKernelTime();

    cudaMemcpy(dev_m1, m1, N * N * sizeof(float), cudaMemcpyHostToDevice);
	cudaMemcpy(dev_m2, m2, N * N * sizeof(float), cudaMemcpyHostToDevice);

	tiledMatrixKernel <<< NUM_THREADS_PER_BLOCK_TILED, NUM_BLOCKS_TILED >>>(dev_m1, dev_m2, dev_result);

	// copy the output to the host
    cudaMemcpy(result, dev_result, N * N * sizeof(float), cudaMemcpyDeviceToHost);

	//stopKernelTime();
	printResults(stop());
    
	// free the device memory
	cudaFree(dev_m1);
	cudaFree(dev_m2);
	cudaFree(dev_result);
}
*/

int main (int argc, char *argv[]) {

    unsigned seed=0;
    float *a = (float*)malloc(sizeof(float)*N*N);
    float *b = (float*)malloc(sizeof(float)*N*N);
    float *c = (float*)malloc(sizeof(float)*N*N);
    
    srand(seed);
    
    //build matrix A with random values and C initilized with 0's
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            *(a+i*N+j) = rand();
            *(c+i*N+j) = 0;
        }   
    }

    //build matrix B with all elements equals to 1
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++)
            *(b+i*N+j) = 1;
    }

	gpuMatrixMult(a,b,c);

	//gpuTiledMatrixMult(a,b,c);

	return 0;
}
